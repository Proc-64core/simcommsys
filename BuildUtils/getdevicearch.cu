#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

// error wrappers

#define cudaSafeCall(error) __cudaSafeCall(error, __FILE__, __LINE__)

inline void __cudaSafeCall(const hipError_t error, const char *file,
      const int line)
   {
   if (error == hipSuccess)
      return;
   std::cerr << "CUDA error in file <" << file << ">, line " << line << " : "
         << hipGetErrorString(error) << ".\n";
   hipDeviceReset();
   exit(1);
   }

// main program

int main()
   {
   int devices = 0;
   cudaSafeCall(hipGetDeviceCount(&devices));

   // check we have at least one device installed
   if (devices == 0)
      {
      std::cerr << "No CUDA-capable devices found" << std::endl;
      exit(1);
      }

   // determine highest compute capability for installed devices
   int max_cc = 0;
   for (int i = 0; i < devices; i++)
      {
      hipDeviceProp_t prop;
      cudaSafeCall(hipGetDeviceProperties(&prop, i));
      const int cc = prop.major * 10 + prop.minor;
      max_cc = std::max(max_cc, cc);
      }

   // print the architecture number
   std::cout << max_cc << std::endl;

   return 0;
   }