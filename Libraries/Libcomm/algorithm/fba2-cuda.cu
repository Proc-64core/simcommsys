#include "hip/hip_runtime.h"
/*!
 * \file
 * 
 * Copyright (c) 2010 Johann A. Briffa
 * 
 * This file is part of SimCommSys.
 *
 * SimCommSys is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SimCommSys is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SimCommSys.  If not, see <http://www.gnu.org/licenses/>.
 * 
 * \section svn Version Control
 * - $Id$
 */

#include "fba2-cuda.h"
#include "pacifier.h"
#include "vectorutils.h"
#include "cuda/gputimer.h"
#include <iomanip>

namespace cuda {

// Determine debug level:
// 1 - Normal debug output only
// 2 - Show allocated memory sizes
// 3 - Show input and intermediate vectors when decoding
#ifndef NDEBUG
#  undef DEBUG
#  define DEBUG 1
#endif

// *** Metric Computer ***

// Internal procedures

// common small tasks

template <class receiver_t, class sig, class real>
__device__
real fba2<receiver_t, sig, real>::metric_computer::get_threshold(const dev_array2r_ref_t& metric, int row, int cols, real factor)
   {
   const bool thresholding = (factor > 0);
   real threshold = 0;
   if (thresholding)
      {
      for (int col = 0; col < cols; col++)
         {
         if (metric(row, col) > threshold)
            {
            threshold = metric(row, col);
            }
         }
      threshold *= factor;
      }
   return threshold;
   }

/*! \brief Returns the sum of the elements in the given array (length N)
 * The sum is computed in parallel between the threads in a given block.
 * A limitation for this to work is that N must be a multiple of 2, and
 * the block size has to be at least N/2 threads.
 * \warning The contents of the array are destroyed in the process.
 */
template <class receiver_t, class sig, class real>
__device__
real fba2<receiver_t, sig, real>::metric_computer::parallel_sum(real array[], const int N)
   {
   const int i = threadIdx.x;
   cuda_assert(N % 2 == 0);
   cuda_assert(N / 2 <= blockDim.x); // Total number of active threads
   for(int n = N; n > 1; n >>= 1)
      {
      const int half = (n >> 1); // divide by two
      // only the first half of the threads will be active.
      if (i < half)
         {
         array[i] += array[i + half];
         }
      // wait until all threads have completed their part
      __syncthreads();
      }
   return array[0];
   }

template <class receiver_t, class sig, class real>
__device__
real fba2<receiver_t, sig, real>::metric_computer::get_scale(const dev_array2r_ref_t& metric, int row, int cols)
   {
   real scale = 0;
   for (int col = 0; col < cols; col++)
      {
      scale += metric(row, col);
      }
   cuda_assertalways(scale > real(0));
   scale = real(1) / scale;
   return scale;
   }

template <class receiver_t, class sig, class real>
__device__
void fba2<receiver_t, sig, real>::metric_computer::normalize(dev_array2r_ref_t& metric, int row, int cols)
   {
   // set up thread index
   const int col = threadIdx.x;
   cuda_assert(col < cols);
   // determine the scale factor to use (each thread has to do this)
   const real scale = get_scale(metric, row, cols);
   // wait until all threads have completed the computation
   __syncthreads();
   // scale all results
   metric(row, col) *= scale;
   }

// decode functions

template <class receiver_t, class sig, class real>
__device__
void fba2<receiver_t, sig, real>::metric_computer::work_gamma_single(const dev_array1s_ref_t& r,
      const dev_array2r_ref_t& app)
   {
   using cuda::min;
   using cuda::max;

   // set up block & thread indexes
   const int i = blockIdx.x;
   const int d = threadIdx.x;
   // compute all matrix values
   // - all threads are independent and indexes guaranteed in range

   // limits on insertions and deletions must be respected:
   //   x2-x1 <= n*I
   //   x2-x1 >= -n
   // limits on introduced drift in this section:
   // (necessary for forward recursion on extracted segment)
   //   x2-x1 <= dxmax
   //   x2-x1 >= -dxmax
   for (int x = -xmax; x <= xmax; x++)
      {
      // clear gamma entries
      for (int deltax = dmin; deltax <= dmax; deltax++)
         {
         gamma(get_gamma_index(d, i, x, deltax)) = 0;
         }
      // limit on end-state (-xmax <= x2 <= xmax):
      //   x2-x1 <= xmax-x1
      //   x2-x1 >= -xmax-x1
      const int deltaxmin = max(-xmax - x, dmin);
      const int deltaxmax = min(xmax - x, dmax);
      for (int deltax = deltaxmin; deltax <= deltaxmax; deltax++)
         {
         gamma(get_gamma_index(d, i, x, deltax)) = compute_gamma_single(d, i, x, deltax, r, app);
         }
      }
   }

template <class receiver_t, class sig, class real>
__device__
void fba2<receiver_t, sig, real>::metric_computer::work_gamma_batch(const dev_array1s_ref_t& r,
      const dev_array2r_ref_t& app)
   {
   using cuda::min;
   using cuda::max;

   // set up block & thread indexes
   const int i = blockIdx.x;
   const int d = threadIdx.x;
   // compute all matrix values
   // - all threads are independent and indexes guaranteed in range

   // set up space for batch results
   libcomm::bsid::real ptable_data[libcomm::bsid::metric_computer::arraysize];
   cuda_assertalways(libcomm::bsid::metric_computer::arraysize >= 2 * dxmax + 1);
   cuda::vector_reference<libcomm::bsid::real> ptable(ptable_data, 2 * dxmax + 1);
   // compute metric with batch interface
   for (int x = -xmax; x <= xmax; x++)
      {
      compute_gamma_batch(d, i, x, ptable, r, app);
      // copy results
      for (int deltax = dmin; deltax <= dmax; deltax++)
         {
         gamma(get_gamma_index(d, i, x, deltax)) = ptable(dxmax + deltax);
         }
      }
   }

template <class receiver_t, class sig, class real>
__device__
void fba2<receiver_t, sig, real>::metric_computer::work_alpha(const dev_array1r_ref_t& sof_prior, int i)
   {
   using cuda::min;
   using cuda::max;

   // local flag for path thresholding
   const bool thresholding = (th_inner > 0);
   // set up block & thread indexes
   const int x2 = blockIdx.x - xmax;
   const int d = threadIdx.x;
   // set up variables shared within block
   SharedMemory<real> shared;
   real* this_alpha = shared.getPointer();

   if(i == 0)
      {
      // set array initial conditions (parallelized):
      if (d == 0)
         {
         // set initial drift distribution
         alpha(0, x2 + xmax) = sof_prior(x2 + xmax);
         }
      }
   else
      {
      // compute remaining matrix values:
      // determine the strongest path at this point
      const real threshold = get_threshold(alpha, i - 1, 2 * xmax + 1, th_inner);
      // initialize result holder
      this_alpha[d] = 0;
      // limits on deltax can be combined as (c.f. allocate() for details):
      //   x2-x1 <= dmax
      //   x2-x1 >= dmin
      const int x1min = max(-xmax, x2 - dmax);
      const int x1max = min(xmax, x2 - dmin);
      for (int x1 = x1min; x1 <= x1max; x1++)
         {
         // cache previous alpha value in a register
         const real prev_alpha = alpha(i - 1, x1 + xmax);
         // ignore paths below a certain threshold
         if (!thresholding || prev_alpha >= threshold)
            {
            // each block computes for a different end-state (x2)
            // each thread in a block is computing for a different symbol (d)
            real temp = prev_alpha;
            temp *= get_gamma(d, i - 1, x1, x2 - x1);
            this_alpha[d] += temp;
            }
         }
      // make sure all threads in block have finished updating this_alpha
      __syncthreads();
      // compute sum of shared array
      const real temp = parallel_sum(this_alpha, q);
      // store result (first thread in block)
      if (d == 0)
         {
         alpha(i, x2 + xmax) = temp;
         }
      }
   }

template <class receiver_t, class sig, class real>
__device__
void fba2<receiver_t, sig, real>::metric_computer::work_beta(const dev_array1r_ref_t& eof_prior, int i)
   {
   using cuda::min;
   using cuda::max;

   // local flag for path thresholding
   const bool thresholding = (th_inner > 0);
   // set up block & thread indexes
   const int x1 = blockIdx.x - xmax;
   const int d = threadIdx.x;
   // set up variables shared within block
   SharedMemory<real> shared;
   real* this_beta = shared.getPointer();

   if(i == N)
      {
      // set array initial conditions (parallelized):
      if (d == 0)
         {
         // set final drift distribution
         beta(N, x1 + xmax) = eof_prior(x1 + xmax);
         }
      }
   else
      {
      // compute remaining matrix values:
      // determine the strongest path at this point
      const real threshold = get_threshold(beta, i + 1, 2 * xmax + 1, th_inner);
      // initialize result holder
      this_beta[d] = 0;
      // limits on deltax can be combined as (c.f. allocate() for details):
      //   x2-x1 <= dmax
      //   x2-x1 >= dmin
      const int x2min = max(-xmax, dmin + x1);
      const int x2max = min(xmax, dmax + x1);
      for (int x2 = x2min; x2 <= x2max; x2++)
         {
         // cache next beta value in a register
         const real next_beta = beta(i + 1, x2 + xmax);
         // ignore paths below a certain threshold
         if (!thresholding || next_beta >= threshold)
            {
            // each block computes for a different start-state (x1)
            // each thread in a block is computing for a different symbol (d)
            real temp = next_beta;
            temp *= get_gamma(d, i, x1, x2 - x1);
            this_beta[d] += temp;
            }
         }
      // make sure all threads in block have finished updating this_beta
      __syncthreads();
      // compute sum of shared array
      const real temp = parallel_sum(this_beta, q);
      // store result
      if (d == 0)
         {
         beta(i, x1 + xmax) = temp;
         }
      }
   }

template <class receiver_t, class sig, class real>
__device__
void fba2<receiver_t, sig, real>::metric_computer::work_message_app(dev_array2r_ref_t& ptable) const
   {
   using cuda::min;
   using cuda::max;

   // local flag for path thresholding
   const bool thresholding = (th_outer > 0);
   // Check result vector (one symbol per timestep)
   cuda_assertalways(ptable.get_rows()==N && ptable.get_cols()==q);
   // set up block & thread indexes
   const int i = blockIdx.x;
   const int d = threadIdx.x;
   // ptable(i,d) is the a posteriori probability of having transmitted symbol 'd' at time 'i'
   // - all threads are independent and indexes guaranteed in range
   // determine the strongest path at this point
   const real threshold = get_threshold(alpha, i, 2 * xmax + 1, th_outer);
   // initialize result holder
   real p = 0;
   for (int x1 = -xmax; x1 <= xmax; x1++)
      {
      // cache this alpha value in a register
      const real this_alpha = alpha(i, x1 + xmax);
      // ignore paths below a certain threshold
      if (!thresholding || this_alpha >= threshold)
         {
         // limits on deltax can be combined as (c.f. allocate() for details):
         //   x2-x1 <= dmax
         //   x2-x1 >= dmin
         const int x2min = max(-xmax, dmin + x1);
         const int x2max = min(xmax, dmax + x1);
         for (int x2 = x2min; x2 <= x2max; x2++)
            {
            real temp = this_alpha;
            temp *= beta(i + 1, x2 + xmax);
            temp *= get_gamma(d, i, x1, x2 - x1);
            p += temp;
            }
         }
      }
   // store result
   ptable(i,d) = p;
   }

template <class receiver_t, class sig, class real>
__device__
void fba2<receiver_t, sig, real>::metric_computer::work_state_app(dev_array1r_ref_t& ptable,
      const int i) const
   {
   // Check result vector and requested index
   cuda_assertalways(ptable.size()==2*xmax+1);
   cuda_assert(i >= 0 && i <= N);
   // set up block & thread indexes
   const int x = threadIdx.x - xmax;
   //const int d = threadIdx.x;
   // compute posterior probabilities for given index
   ptable(x + xmax) = alpha(i, x + xmax) * beta(i, x + xmax);
   }

// Kernels
// NOTE: these *must* be global functions

template <class receiver_t, class sig, class real>
__global__
void fba2_gamma_single_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, const vector_reference<sig> r,
      const matrix_reference<real> app)
   {
   object().work_gamma_single(r, app);
   }

template <class receiver_t, class sig, class real>
__global__
void fba2_gamma_batch_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, const vector_reference<sig> r,
      const matrix_reference<real> app)
   {
   object().work_gamma_batch(r, app);
   }

template <class receiver_t, class sig, class real>
__global__
void fba2_alpha_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, const vector_reference<real> sof_prior, const int i)
   {
   object().work_alpha(sof_prior, i);
   }

template <class receiver_t, class sig, class real>
__global__
void fba2_normalize_alpha_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, const int i)
   {
   object().normalize_alpha(i);
   }

template <class receiver_t, class sig, class real>
__global__
void fba2_beta_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, const vector_reference<real> eof_prior, const int i)
   {
   object().work_beta(eof_prior, i);
   }

template <class receiver_t, class sig, class real>
__global__
void fba2_normalize_beta_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, const int i)
   {
   object().normalize_beta(i);
   }

template <class receiver_t, class sig, class real>
__global__
void fba2_message_app_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, matrix_reference<real> ptable)
   {
   object().work_message_app(ptable);
   }

template <class receiver_t, class sig, class real>
__global__
void fba2_state_app_kernel(value_reference<typename fba2<receiver_t, sig, real>::metric_computer> object, vector_reference<real> ptable, const int i)
   {
   object().work_state_app(ptable, i);
   }

// *** Main Class

// Memory allocation

/*! \brief Memory allocator for working matrices
 */
template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::allocate()
   {
   // flag the state of the arrays
   initialised = true;

   // determine allowed limits on deltax:
   // limits on insertions and deletions:
   //   x2-x1 <= n*I
   //   x2-x1 >= -n
   // limits on introduced drift in this section:
   // (necessary for forward recursion on extracted segment)
   //   x2-x1 <= dxmax
   //   x2-x1 >= -dxmax
   // the above two sets of limits can be combined as:
   //   x2-x1 <= min(n*I, dxmax) = dmax
   //   x2-x1 >= max(-n, -dxmax) = dmin
   computer.dmin = std::max(-computer.n, -computer.dxmax);
   computer.dmax = std::min(computer.n * computer.I, computer.dxmax);
   // alpha needs indices (i,x) where i in [0, N] and x in [-xmax, xmax]
   // beta needs indices (i,x) where i in [0, N] and x in [-xmax, xmax]
   alpha.init(computer.N + 1, 2 * computer.xmax + 1); // offsets: 0, xmax
   beta.init(computer.N + 1, 2 * computer.xmax + 1); // offsets: 0, xmax

   // gamma needs indices (d,i,x,deltax) where d in [0, q-1], i in [0, N-1]
   // x in [-xmax, xmax], and deltax in [dmin, dmax] = [max(-n,-dxmax), min(nI,dxmax)]
   // (note: this is allocated as a flat sequence)
   if (computer.flags.globalstore)
      {
      gamma.init(computer.q * computer.N * (2 * computer.xmax + 1)
            * (computer.dmax - computer.dmin + 1));
      }
   else
      {
      gamma.init(0);
      }
   // need to keep track only if we're globally storing lazy computations
   // cached needs indices (i,x) where i in [0, N-1] and x in [-xmax, xmax]
   if (computer.flags.lazy && computer.flags.globalstore)
      {
      cached.init(computer.N, 2 * computer.xmax + 1); // offsets: 0, xmax
      }
   else
      {
      cached.init(0, 0);
      }
   // copy over to references
   computer.alpha = alpha;
   computer.beta = beta;
   computer.gamma = gamma;
   computer.cached = cached;

   // if this is not the first time, skip the rest
   static bool first_time = true;
   if (!first_time)
      return;
   first_time = false;

#ifndef NDEBUG
   // set required format, storing previous settings
   const std::ios::fmtflags flags = std::cerr.flags();
   std::cerr.setf(std::ios::fixed, std::ios::floatfield);
   const int prec = std::cerr.precision(1);
   // determine memory occupied and tell user
   const size_t bytes_used = sizeof(bool) * cached.size() + sizeof(real)
         * (alpha.size() + beta.size() + gamma.size());
   std::cerr << "FBA Memory Usage: " << bytes_used / double(1 << 20) << "MiB"
         << std::endl;
   // revert cerr to original format
   std::cerr.precision(prec);
   std::cerr.flags(flags);
#endif

#ifndef NDEBUG
   // determine required space for inner metric table (Jiao-Armand method)
   size_t entries = 0;
   for (int delta = computer.dmin; delta <= computer.dmax; delta++)
      entries += (1 << (delta + computer.n));
   std::cerr << "Jiao-Armand Table Size: " << computer.q * entries
         * sizeof(float) / double(1 << 20) << "MiB" << std::endl;
#endif

#if DEBUG>=2
   std::cerr << "Allocated FBA memory..." << std::endl;
   std::cerr << "dmax = " << computer.dmax << std::endl;
   std::cerr << "dmin = " << computer.dmin << std::endl;
   std::cerr << "alpha = " << computer.N + 1 << "x" << 2 * computer.xmax + 1
   << " = " << alpha.size() << std::endl;
   std::cerr << "beta = " << computer.N + 1 << "x" << 2 * computer.xmax + 1
   << " = " << beta.size() << std::endl;
   std::cerr << "gamma = " << computer.q << "x" << computer.N << "x" << 2
   * computer.xmax + 1 << "x" << computer.dmax - computer.dmin + 1
   << " = " << gamma.size() << std::endl;
#endif
   }

/*! \brief Release memory for working matrices
 */
template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::free()
   {
   alpha.init(0, 0);
   beta.init(0, 0);
   gamma.init(0);
   cached.init(0, 0);
   // copy over to references
   computer.alpha = alpha;
   computer.beta = beta;
   computer.gamma = gamma;
   computer.cached = cached;
   // flag the state of the arrays
   initialised = false;
   }

// helper methods

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::reset_cache() const
   {
   // initialise array
   gamma.fill(0);
   // initialize cache
   cached.fill(false);
   }

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::print_gamma(std::ostream& sout) const
   {
   // copy the data set from the device
   libbase::vector<real> host_gamma = libbase::vector<real>(gamma);
   // gamma has indices (d,i,x,deltax) where:
   //    d in [0, q-1], i in [0, N-1], x in [-xmax, xmax], and
   //    deltax in [dmin, dmax] = [max(-n,-xmax), min(nI,xmax)]
   for (int i = 0; i < computer.N; i++)
      {
      sout << "i = " << i << ":" << std::endl;
      for (int d = 0; d < computer.q; d++)
         {
         sout << "d = " << d << ":" << std::endl;
         for (int x = -computer.xmax; x <= computer.xmax; x++)
            {
            for (int deltax = computer.dmin; deltax <= computer.dmax; deltax++)
               {
               const int ndx = computer.get_gamma_index(d, i, x, deltax);
               sout << '\t' << host_gamma(ndx);
               }
            sout << std::endl;
            }
         }
      }
   }

// data movement

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::copy_table(const dev_array2r_t& dev_table,
      array1vr_t& table)
   {
   // determine source sizes
   const int rows = dev_table.get_rows();
   const int cols = dev_table.get_cols();
   // initialise result table and copy one row at a time
   libbase::allocate(table, rows, cols);
   for (int i = 0; i < rows; i++)
      {
      table(i) = array1r_t(dev_table.extract_row(i));
      }
   }

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::copy_table(const array1vd_t& table,
      dev_array2r_t& dev_table)
   {
   // determine source sizes
   const int rows = table.size();
   const int cols = (rows > 0) ? table(0).size() : 0;
   // initialise result table and copy one row at a time
   dev_table.init(rows, cols);
   for (int i = 0; i < rows; i++)
      {
      assert(table(i).size() == cols);
      dev_table.extract_row(i) = array1r_t(table(i));
      }
   }

// de-reference kernel calls

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::work_gamma(const dev_array1s_t& r,
      const dev_array2r_t& app)
   {
   assert( initialised);
   // Shorthand
   const int N = computer.N;
   const int q = computer.q;
   // Gamma computation:
   if (computer.flags.lazy)
      {
      // keep a copy of received vector and a-priori statistics
      computer.r = dev_r;
      computer.app = dev_app;
      // re-create a copy of the device object (to pass to kernels)
      dev_object = computer;
      // reset cache values if we're using it
      if (computer.flags.globalstore)
         reset_cache();
      }
   else
      {
      // inform user what the kernel sizes are
      static bool first_time = true;
      if (first_time)
         {
         std::cerr << "Gamma Kernel: " << N << " blocks x " << q << " threads"
               << std::endl;
         first_time = false;
         }
      // pre-computation
      if (computer.flags.batch)
         {
         // block index is for i in [0, N-1]: grid size = N
         // thread index is for d in [0, q-1]: block size = q
         fba2_gamma_batch_kernel<receiver_t, sig, real> <<<N,q>>>(dev_object, r, app);
         cudaSafeThreadSynchronize();
         }
      else
         {
         // block index is for i in [0, N-1]: grid size = N
         // thread index is for d in [0, q-1]: block size = q
         fba2_gamma_single_kernel<receiver_t, sig, real> <<<N,q>>>(dev_object, r, app);
         cudaSafeThreadSynchronize();
         }
      }
   }

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::work_alpha(const dev_array1r_t& sof_prior)
   {
   assert( initialised);
   // Shorthand
   const int N = computer.N;
   const int q = computer.q;
   const int xmax = computer.xmax;
   // inform user what the kernel sizes are
   static bool first_time = true;
   if (first_time)
      {
      std::cerr << "Alpha Kernel: " << 2 * xmax + 1 << " blocks x " << q
            << " threads" << std::endl;
      if (computer.flags.norm)
         {
         std::cerr << "Normalization Kernel: " << 1 << " blocks x " << 2 * xmax
               + 1 << " threads" << std::endl;
         }
      first_time = false;
      }
   // Alpha computation:
   for (int i = 0; i <= N; i++)
      {
      // block index is for x2 in [-xmax, xmax]: grid size = 2*xmax+1
      // thread index is for d in [0, q-1]: block size = q
      // shared memory: array of q 'real's
      fba2_alpha_kernel<receiver_t, sig, real> <<<2*xmax+1,q,q*sizeof(real)>>>(dev_object, sof_prior, i);
      cudaSafeThreadSynchronize();
      // normalize if requested
      if (computer.flags.norm)
         {
         // NOTE: this has to be done in one block, as we need to sync after
         //       determining the scale to use 
         // block index is not used: grid size = 1
         // thread index is for x2 in [-xmax, xmax]: block size = 2*xmax+1
         fba2_normalize_alpha_kernel <receiver_t, sig, real> <<<1,2*xmax+1>>>(dev_object, i);
         cudaSafeThreadSynchronize();
         }
      }
   }

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::work_beta(const dev_array1r_t& eof_prior)
   {
   assert( initialised);
   // Shorthand
   const int N = computer.N;
   const int q = computer.q;
   const int xmax = computer.xmax;
   // inform user what the kernel sizes are
   static bool first_time = true;
   if (first_time)
      {
      std::cerr << "Beta Kernel: " << 2 * xmax + 1 << " blocks x " << q
            << " threads" << std::endl;
      if (computer.flags.norm)
         {
         std::cerr << "Normalization Kernel: " << 1 << " blocks x " << 2 * xmax
               + 1 << " threads" << std::endl;
         }
      first_time = false;
      }
   // Beta computation:
   for (int i = N; i >= 0; i--)
      {
      // block index is for x2 in [-xmax, xmax]: grid size = 2*xmax+1
      // thread index is for d in [0, q-1]: block size = q
      // shared memory: array of q 'real's
      fba2_beta_kernel<receiver_t, sig, real> <<<2*xmax+1,q,q*sizeof(real)>>>(dev_object, eof_prior, i);
      cudaSafeThreadSynchronize();
      // normalize if requested
      if (computer.flags.norm)
         {
         // NOTE: this has to be done in one block, as we need to sync after
         //       determining the scale to use 
         // block index is not used: grid size = 1
         // thread index is for x2 in [-xmax, xmax]: block size = 2*xmax+1
         fba2_normalize_beta_kernel <receiver_t, sig, real> <<<1,2*xmax+1>>>(dev_object, i);
         cudaSafeThreadSynchronize();
         }
      }
   }

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::work_results(dev_array2r_t& ptable,
      dev_array1r_t& sof_post, dev_array1r_t& eof_post) const
   {
   assert( initialised);
   // Shorthand
   const int N = computer.N;
   const int q = computer.q;
   const int xmax = computer.xmax;
   // inform user what the kernel sizes are
   static bool first_time = true;
   if (first_time)
      {
      std::cerr << "Message APP Kernel: " << N << " blocks x " << q
            << " threads" << std::endl;
      std::cerr << "State APP Kernel (x2): " << 1 << " blocks x " << 2 * xmax
            + 1 << " threads" << std::endl;
      first_time = false;
      }
   // Results computation:
   // compute APPs of message
   // block index is for i in [0, N-1]: grid size = N
   // thread index is for d in [0, q-1]: block size = q
   fba2_message_app_kernel<receiver_t, sig, real> <<<N,q>>>(dev_object, ptable);
   cudaSafeThreadSynchronize();
   // compute APPs of sof/eof state values 
   // block index is not used: grid size = 1
   // thread index is for x in [-xmax, xmax]: block size = 2*xmax+1
   fba2_state_app_kernel<receiver_t, sig, real> <<<1,2*xmax+1>>>(dev_object, sof_post, 0);
   cudaSafeThreadSynchronize();
   fba2_state_app_kernel<receiver_t, sig, real> <<<1,2*xmax+1>>>(dev_object, eof_post, N);
   cudaSafeThreadSynchronize();
   }

// User procedures

// Initialization

template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::init(int N, int n, int q, int I, int xmax,
      int dxmax, double th_inner, double th_outer, bool norm, bool batch,
      bool lazy, bool globalstore)
   {
   // if any parameters that effect memory have changed, release memory
   if (initialised && (N != computer.N || n != computer.n || q != computer.q
         || I != computer.I || xmax != computer.xmax || dxmax != computer.dxmax
         || lazy != computer.flags.lazy || globalstore
         != computer.flags.globalstore))
      {
      free();
      }
   // code parameters
   assert(N > 0);
   assert(n > 0);
   computer.N = N;
   computer.n = n;
   assert(q > 1);
   computer.q = q;
   // decoder parameters
   assert(I > 0);
   assert(xmax > 0);
   assert(dxmax > 0);
   computer.I = I;
   computer.xmax = xmax;
   computer.dxmax = dxmax;
   // path truncation parameters
   assert(th_inner >= 0 && th_inner <= 1);
   assert(th_outer >= 0 && th_outer <= 1);
   computer.th_inner = th_inner;
   computer.th_outer = th_outer;
   // decoding mode parameters
   assertalways(lazy || globalstore); // pre-compute without global storage not yet supported
   computer.flags.norm = norm;
   computer.flags.batch = batch;
   computer.flags.lazy = lazy;
   computer.flags.globalstore = globalstore;
   }

/*!
 * \brief Frame decode cycle
 * \param[in] collector Reference to (instrumented) results collector object
 * \param[in] r Received frame
 * \param[in] sof_prior Prior probabilities for start-of-frame position
 *                      (zero-index matches zero-index of r)
 * \param[in] eof_prior Prior probabilities for end-of-frame position
 *                      (zero-index matches tau-index of r, where tau is the
 *                      length of the transmitted frame)
 * \param[in] app A-Priori Probabilities for message
 * \param[out] ptable Posterior Probabilities for message
 * \param[out] sof_post Posterior probabilities for start-of-frame position
 *                      (indexing same as prior)
 * \param[out] eof_post Posterior probabilities for end-of-frame position
 *                      (indexing same as prior)
 * \param[in] offset Index offset for prior, post, and r vectors
 *
 * \note If APP table is empty, it is assumed that symbols are equiprobable.
 *
 * \note Priors for start and end-of-frame *must* be supplied; in the case of a
 *       received frame with exactly known boundaries, this must be offset by
 *       xmax and padded to a total length of tau + 2*xmax, where tau is the
 *       length of the transmitted frame. This avoids special handling for such
 *       vectors.
 *
 * \note Offset is the same as for stream_modulator.
 */
template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::decode(libcomm::instrumented& collector,
      const array1s_t& r, const array1d_t& sof_prior,
      const array1d_t& eof_prior, const array1vd_t& app, array1vr_t& ptable,
      array1r_t& sof_post, array1r_t& eof_post, const int offset)
   {
#if DEBUG>=3
   std::cerr << "Starting decode..." << std::endl;
   std::cerr << "N = " << computer.N << std::endl;
   std::cerr << "n = " << computer.n << std::endl;
   std::cerr << "q = " << computer.q << std::endl;
   std::cerr << "I = " << computer.I << std::endl;
   std::cerr << "xmax = " << computer.xmax << std::endl;
   std::cerr << "dxmax = " << computer.dxmax << std::endl;
   std::cerr << "th_inner = " << computer.th_inner << std::endl;
   std::cerr << "th_outer = " << computer.th_outer << std::endl;
   std::cerr << "norm = " << computer.flags.norm << std::endl;
   std::cerr << "real = " << typeid(real).name() << std::endl;
#endif
   // Initialise memory on device if necessary
   if (!initialised)
      allocate();
   // Validate sizes and offset
   const int tau = computer.N * computer.n;
   assertalways(offset == computer.xmax);
   assertalways(r.size() == tau + 2 * computer.xmax);
   assertalways(sof_prior.size() == 2 * computer.xmax + 1);
   assertalways(eof_prior.size() == 2 * computer.xmax + 1);

   // Setup device
   gputimer ts("t_setup");
   // copy input data to device, allocating space as needed
   dev_r = r;
   copy_table(app, dev_app);
   dev_sof_table = array1r_t(sof_prior);
   dev_eof_table = array1r_t(eof_prior);
   // allocate space on device for result
   dev_ptable.init(computer.N, computer.q);
   // create a copy of the device object (to pass to kernels)
   dev_object = computer;
   collector.add_timer(ts);
#if DEBUG>=3
   // show input data, as on device
   std::cerr << "r = " << array1s_t(dev_r) << std::endl;
   std::cerr << "app = " << array2r_t(dev_app) << std::endl;
   std::cerr << "sof_prior = " << array1r_t(dev_sof_table) << std::endl;
   std::cerr << "eof_prior = " << array1r_t(dev_eof_table) << std::endl;
#endif

   // Gamma
   gputimer tg("t_gamma");
   work_gamma(dev_r, dev_app);
   collector.add_timer(tg);
#if DEBUG>=3
   if (!computer.flags.lazy && computer.flags.globalstore)
      {
      std::cerr << "gamma = " << std::endl;
      print_gamma(std::cerr);
      }
#endif
   // Alpha + Beta
   gputimer tab("t_alpha+beta");
   // Alpha
   gputimer ta("t_alpha");
   work_alpha( dev_sof_table);
   collector.add_timer(ta);
#if DEBUG>=3
   std::cerr << "alpha = " << libbase::matrix<real>(alpha) << std::endl;
#endif
   // Beta
   gputimer tb("t_beta");
   work_beta( dev_eof_table);
   collector.add_timer(tb);
   collector.add_timer(tab);
#if DEBUG>=3
   std::cerr << "beta = " << libbase::matrix<real>(beta) << std::endl;
#endif
   // Results computation
   gputimer tr("t_results");
   work_results(dev_ptable, dev_sof_table, dev_eof_table);
   collector.add_timer(tr);
   // Results transfer
   gputimer tc("t_transfer");
   copy_table(dev_ptable, ptable);
   sof_post = array1r_t(dev_sof_table);
   eof_post = array1r_t(dev_eof_table);
   collector.add_timer(tc);
#if DEBUG>=3
   // show output data
   std::cerr << "ptable = " << ptable << std::endl;
   std::cerr << "sof_post = " << sof_post << std::endl;
   std::cerr << "eof_post = " << eof_post << std::endl;
#endif

   // add values for limits that depend on channel conditions
   collector.add_timer(computer.I, "c_I");
   collector.add_timer(computer.xmax, "c_xmax");
   collector.add_timer(computer.dxmax, "c_dxmax");
   // add memory usage
   collector.add_timer(sizeof(real) * alpha.size(), "m_alpha");
   collector.add_timer(sizeof(real) * beta.size(), "m_beta");
   collector.add_timer(sizeof(real) * gamma.size(), "m_gamma");
   }

/*!
 * \brief Get the posterior channel drift pdf at codeword boundaries
 * \param[out] pdftable Posterior Probabilities for codeword boundaries
 *
 * Codeword boundaries are taken to include frame boundaries, such that
 * pdftable(i) corresponds to the boundary between codewords 'i' and 'i+1'.
 * This method must be called after a call to decode(), so that it can return
 * posteriors for the last transmitted frame.
 */
template <class receiver_t, class sig, class real>
void fba2<receiver_t, sig, real>::get_drift_pdf(array1vr_t& pdftable) const
   {
   assert( initialised);
   // Shorthand
   const int N = computer.N;
   const int q = computer.q;
   const int xmax = computer.xmax;
   // inform user what the kernel sizes are
   static bool first_time = true;
   if (first_time)
      {
      std::cerr << "State APP Kernel (x" << N + 1 << "): " << 1 << " blocks x "
            << 2 * xmax + 1 << " threads" << std::endl;
      first_time = false;
      }
   // Drift PDF computation:
   // allocate space for results
   pdftable.init(N + 1);
   // consider each time index in the order given
   for (int i = 0; i <= N; i++)
      {
      // block index is not used: grid size = 1
      // thread index is for x in [-xmax, xmax]: block size = 2*xmax+1
      fba2_state_app_kernel<receiver_t, sig, real> <<<1,2*xmax+1>>>(dev_object, dev_sof_table, i);
      cudaSafeThreadSynchronize();
      // copy result from temporary space
      pdftable(i) = array1r_t(dev_sof_table);
      }
   }

} // end namespace

// Explicit Realizations

#include "modem/dminner2-receiver-cuda.h"

namespace cuda {

// Explicit Realizations
#include <boost/preprocessor/seq/for_each.hpp>
#include <boost/preprocessor/seq/for_each_product.hpp>
#include <boost/preprocessor/seq/enum.hpp>

#define REAL_TYPE_SEQ \
   (float)(double)

// *** Instantiations for dminner2: bool only ***

#define INSTANTIATE_DM(r, x, type) \
      template class fba2<dminner2_receiver<type> , bool, type> ; \
      template class value<fba2<dminner2_receiver<type> , bool, type>::metric_computer> ;

BOOST_PP_SEQ_FOR_EACH(INSTANTIATE_DM, x, REAL_TYPE_SEQ)

// *** Instantiations for tvb: gf types only ***

//#define INSTANTIATE_TVB(r, args) \
//      template class fba2<tvb_receiver<BOOST_PP_SEQ_ENUM(args)> , \
//         BOOST_PP_SEQ_ENUM(args)> ;
//
//BOOST_PP_SEQ_FOR_EACH_PRODUCT(INSTANTIATE_TVB, (GF_TYPE_SEQ)(REAL_TYPE_SEQ))

} // end namespace
